#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "json.hpp"

using json = nlohmann::json;

// This kernel creates bank conflicts ranging from 0 to 31 per warp for stores.
// Parameter "conflicts" controls the number of conflicts to be simulated.
// If conflicts == 0, then the code just creates wavefronts without conflicts.
__global__ void microben(int *arr, long long int *ticks, long long int *start, long long int *end, long long int *t_ld, long long int *t_st, int conflicts)
{
  __shared__ int s[8192];
  int var, var1, var2;
  int new_var, new_var1, new_var2;
  int addr, addr1, addr2;
  int th = threadIdx.x;
  long long int t1, t2, t_temp;
  //s[th] = arr[th];
  //__syncthreads();
  addr = th % 32;
  addr1 = th % 32;
  if(conflicts != 0) {
    addr = addr % (conflicts + 1);
    addr = addr * 32;
  }
  __syncthreads();
  //__syncthreads();
  t1 = clock64();
  s[addr] = th;
  //__threadfence_block();
  //__syncthreads();
  //s[addr2] = th;
  //__syncthreads();
  //__threadfence_block();
  //s[addr1] = th;
  //__syncthreads();
  //s[addr1+64] = th;
  //__threadfence_block();
  __syncthreads();
  t_temp = clock64();
  //t1 = clock64();
  //t2 = clock64();
  var2 = s[addr1];
  var2 = var2 + 1;
  t2 = clock64();
  //var = s[addr];
  start[th] = t1;
  end[th] = t2;
  t_ld[th] = t2 - t_temp;
  t_st[th] = t_temp - t1;
  ticks[th] = t2-t1;
  arr[th] = var2;
}

int main(int argc, char** argv)
{
    const int threads = atoi(argv[1]);
    const int conflicts = atoi(argv[2]);

    const int n = 8192;
    std::vector <long long int> h_start, h_end, h_time, h_ld, h_st;
    h_start = std::vector<long long int>(threads, std::numeric_limits<long long int>::max());
    h_end = std::vector<long long int>(threads, std::numeric_limits<long long int>::min());
    h_time = std::vector<long long int>(threads, std::numeric_limits<long long int>::min());
    h_ld = std::vector<long long int>(threads, std::numeric_limits<long long int>::max());
    h_st = std::vector<long long int>(threads, std::numeric_limits<long long int>::max());
  
    int h_A[n], h_B[n];
    int *d_A;
    long long int *d_time, *d_start, *d_end, *d_ld, *d_st;
    
    for (int i = 0; i < n; i++) {
        h_A[i] = i;
        h_B[i] = 0;
    }
    
    hipMalloc(&d_A, n * sizeof(int)); 
    hipMalloc(&d_time, threads * sizeof(long long int));
    hipMalloc(&d_start, threads * sizeof(long long int));
    hipMalloc(&d_ld, threads * sizeof(long long int));
    hipMalloc(&d_st, threads * sizeof(long long int));
    hipMalloc(&d_end, threads * sizeof(long long int));
    
    hipMemcpy(d_A, h_A, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_time, h_time.data(), threads*sizeof(long long int), hipMemcpyHostToDevice);
    hipMemcpy(d_start, h_start.data(), threads*sizeof(long long int), hipMemcpyHostToDevice);
    hipMemcpy(d_ld, h_ld.data(), threads*sizeof(long long int), hipMemcpyHostToDevice);
    hipMemcpy(d_st, h_st.data(), threads*sizeof(long long int), hipMemcpyHostToDevice);
    hipMemcpy(d_end, h_end.data(), threads*sizeof(long long int), hipMemcpyHostToDevice);

    //microben1<<<1,threads>>>(d_A, d_time, d_start, d_end, conflicts);
    microben<<<1,threads>>>(d_A, d_time, d_start, d_end, d_ld, d_st, conflicts);

    hipMemcpy(h_B, d_A, n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_time.data(), d_time, threads*sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(h_start.data(), d_start, threads*sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(h_ld.data(), d_ld, threads*sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(h_st.data(), d_st, threads*sizeof(long long int), hipMemcpyDeviceToHost);
    hipMemcpy(h_end.data(), d_end, threads*sizeof(long long int), hipMemcpyDeviceToHost);

    long long int time = *std::max_element(h_end.cbegin(), h_end.cend()) - *std::min_element(h_start.cbegin(), h_start.cend());
    //long long int t_ld = *std::max_element(h_ld.cbegin(), h_ld.cend()) - *std::min_element(h_ld.cbegin(), h_ld.cend());
    //long long int t_st = *std::max_element(h_st.cbegin(), h_st.cend()) - *std::min_element(h_st.cbegin(), h_st.cend());
    nlohmann::json obj;
    const int warps = ceil(float(threads)/32.0);
    obj["warps"] = warps;
    obj["Conflicts"] = conflicts;
    // obj["wid"] = nlohmann::json::array();
    // obj["thid"] = nlohmann::json::array();
    // obj["tic"] = nlohmann::json::array();
    // obj["toc"] = nlohmann::json::array();
    // obj["cycles"] = nlohmann::json::array();
    obj["T"] = time;

    long long int basevalue = *std::min_element(h_start.cbegin(), h_start.cend());
    //for(int i=0; i < threads; i++){
    //    std::cout << "Thread " << i << " Start: " << (h_start[i]-basevalue) << " End:" << (h_end[i]-basevalue) << " Load: " << h_ld[i] << " Store: " << h_st[i] << " Clock_Cycles: " <<  h_time[i] << std::endl;
        // obj["wid"].push_back(i/32);
        // obj["thid"].push_back(i);
        // obj["tic"].push_back(h_start[i]-basevalue);
        // obj["toc"].push_back(h_end[i]-basevalue);
        // obj["cycles"].push_back(h_time[i]);
    //}

    //std::cout << obj << std::endl;
    //std::cout << warps << "," << conflicts << "," << time << std::endl;
    std::cout << time << std::endl;

    hipFree(d_A);
    hipFree(d_time);
    hipFree(d_start);
    hipFree(d_end);
    hipFree(d_ld);
    hipFree(d_st);

    return 0;
}